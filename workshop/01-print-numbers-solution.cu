
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printNumber(int number)
{
  printf("%d\n", number);
}

int main()
{
  hipStream_t stream[5];       // CUDA streams are of type `cudaStream_t`.
  for(int i = 0; i < 5; i++)
     hipStreamCreate(&stream[i]); 
  for (int i = 0; i < 5; ++i)
  {
    printNumber<<<1, 1, 0, stream[i]>>>(i);
  }
  hipDeviceSynchronize();
  for (int i = 0; i < 5; ++i)
     hipStreamDestroy(stream[i]);
}

