
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N  64

__global__ void matrixMulGPU( int * a, int * b, int * c )
{
  /*
   * Build out this kernel.
   */
   int val = 0;
   int i_x = blockIdx.x * blockDim.x + threadIdx.x;
   int i_y = blockIdx.y * blockDim.y + threadIdx.y;
   int s_x = gridDim.x * blockDim.x;
   int s_y = gridDim.y * blockDim.y;
   for( int row = i_x; row < N; row+=s_x )
     for( int col = i_y; col < N; col+=s_y )
     {
        val = 0;
        for ( int k = i_x; k < N; k+=s_x )
          val += a[row * N + k] * b[k * N + col];
        c[row * N + col] = val;
     }
}

/*
 * This CPU function already works, and will run to create a solution matrix
 * against which to verify your work building out the matrixMulGPU kernel.
 */

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;
  int ind = 0;
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
      //if (ind < 5){
         printf("c[row:%d * N:%d + col:%d] = val:%d\n", row,N,col,val);
      //}
      ind++;
    }
}

int main()
{
  int *a, *b, *c_cpu, *c_gpu; // Allocate a solution matrix for both the CPU and the GPU operations

  int size = N * N * sizeof (int); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  // Initialize memory; create 2D matrices
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
    }

  /*
   * Assign `threads_per_block` and `number_of_blocks` 2D values
   * that can be used in matrixMulGPU above.
   */

  dim3 threads_per_block(4, 4, 1);
  dim3 number_of_blocks(4, 4, 1);

  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );

  hipDeviceSynchronize();

  // Call the CPU version to check our work
  matrixMulCPU( a, b, c_cpu );

  // Compare the two answers to make sure they are equal
  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d], values: c_cpu:%d and c_gpu:%d\n", 
            row, col,
            c_cpu[row * N + col],
            c_gpu[row * N + col]
            );
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  // Free all our allocated memory
  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
}

